#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include <GL/glew.h>
#include <GL/glut.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#define N 10
#define T 27

using namespace std;

__global__ void alpha(hipPitchedPtr devicePitchedPointer)
{   int ix =  blockIdx.x*blockDim.x+threadIdx.x;
    int iy =  blockIdx.y*blockDim.y+threadIdx.y;
    int iz =  blockIdx.z*blockDim.z+threadIdx.z;

    // Get attributes from device pitched pointer
    char     *devicePointer  =   (char *)devicePitchedPointer.ptr;
    size_t    pitch          =   devicePitchedPointer.pitch;
    size_t    slicePitch     =   pitch * 3; //dimension y

    char *current_slice = devicePointer + iz * slicePitch;
    float *current_row = (float*)(current_slice + iy * pitch);
    current_row[ix] = current_row[ix] + 1;
}

int main(void)
{
    // Set up test data
    float image_data[3][3][3] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26};
    // Allocate 3D memory on the device
    hipExtent volumeSizeBytes = make_hipExtent(sizeof(float) * N, N, N);
    hipPitchedPtr devicePitchedPointer;
    hipMalloc3D(&devicePitchedPointer, volumeSizeBytes);

    hipMemcpy3DParms p0 = { 0 }; 
    p0.srcPtr.ptr = image_data;
    p0.srcPtr.pitch = 3 * sizeof(float);
    p0.srcPtr.xsize = 3;
    p0.srcPtr.ysize = 3;
    p0.dstPtr.ptr = devicePitchedPointer.ptr;
    p0.dstPtr.pitch = devicePitchedPointer.pitch;
    p0.dstPtr.xsize = 3;
    p0.dstPtr.ysize = 3;
    p0.extent.width = 3 * sizeof(float);
    p0.extent.height = 3;
    p0.extent.depth = 3;
    p0.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&p0);

    // Kernel Launch Configuration
    dim3 threads_per_block = dim3(3, 3, 3);
    dim3 blocks_per_grid = dim3(1, 1, 1);
    alpha<<<blocks_per_grid, threads_per_block>>>(devicePitchedPointer);

    p0.srcPtr.ptr = devicePitchedPointer.ptr;
    p0.srcPtr.pitch = devicePitchedPointer.pitch;
    p0.dstPtr.ptr = image_data;
    p0.dstPtr.pitch = 3 * sizeof(float); 
    p0.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&p0);
    
    for(int i=0; i < 3; i++){
        for(int j=0; j < 3; j++){
            for(int k=0; k < 3; k++){
                cout << image_data[i][j][k] << endl;
            }
        }
    }
    hipFree(&devicePitchedPointer.ptr);
}